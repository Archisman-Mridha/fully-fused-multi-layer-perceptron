
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sayHelloWorld() { printf("HELLO WORLD from a Kernel"); }

__host__ int main() {
  /*
    The CUDA programming model is a heterogeneous model in which both the CPU and GPU are used.
    In CUDA :

      (1) the host refers to the CPU and its memory (RAM).

      (2) while the device refers to the GPU and its memory (vRAM).

    Code run on the host (CPU) can manage memory on both the host and device, and also launches
    kernels which are functions executed on the device (GPU). These kernels are executed by many
    GPU threads in parallel.

    Given the heterogeneous nature of the CUDA programming model, a typical sequence of operations
    for a CUDA C program is:

      (1) Declare and allocate host and device memory.
      (2) Initialize host data.
      (3) Transfer data from the host to the device.
      (4) Execute one or more kernels.
      (5) Transfer results from the device to the host.
  */

  /*
    <<<1, 1>>> is called the execution configuration, which dictates how many device threads
    execute the kernel in parallel.

    Configuration arguments :

      (1) Number of thread blocks

        A thread block is a programming abstraction that represents a group of threads that can be
        executed in parallel. For better process and data mapping, threads are grouped into thread
        blocks.

      (2) Number of threads per thread block
  */
  sayHelloWorld<<<1, 1>>>();

  return 0;
}
